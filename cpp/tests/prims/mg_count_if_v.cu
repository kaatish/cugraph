#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/high_res_clock.h>
#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/partition_manager.hpp>

#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/prims/any_of_adj_matrix_row.cuh>
#include <cugraph/prims/copy_to_adj_matrix_row_col.cuh>
#include <cugraph/prims/copy_v_transform_reduce_in_out_nbr.cuh>
#include <cugraph/prims/count_if_e.cuh>
#include <cugraph/prims/count_if_v.cuh>
#include <cugraph/prims/reduce_v.cuh>
#include <cugraph/prims/transform_reduce_v.cuh>
#include <cugraph/utilities/error.hpp>
#include <cugraph/vertex_partition_device.cuh>

#include <raft/comms/comms.hpp>
#include <raft/comms/mpi_comms.hpp>
#include <raft/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <gtest/gtest.h>

#include <random>

// do the perf measurements
// enabled by command line parameter s'--perf'
//
static int PERF = 0;

template <typename input_usecase_t>
class Tests_MG_CountIfV
  : public ::testing::TestWithParam<input_usecase_t> {
 public:
  Tests_MG_CountIfV() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of running Katz Centrality on multiple GPUs to that of a single-GPU run
  template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
  void run_current_test(input_usecase_t const &input_usecase)
  {
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto &comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) { --row_comm_size; }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, true, true> mg_graph(handle);
    rmm::device_uvector<vertex_t> d_mg_renumber_map_labels(0, handle.get_stream());
    std::tie(mg_graph, d_mg_renumber_map_labels) =
      input_usecase.template construct_graph<vertex_t, edge_t, weight_t, true, true>(
        handle, true, true);

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();

    std::cout<<"Number of local vertices : "<<mg_graph_view.get_number_of_local_vertices()<<"\n";
    std::cout<<"Number of map labels : "<<d_mg_renumber_map_labels.size()<<"\n";

    vertex_t const * data = d_mg_renumber_map_labels.data();

    auto num_vertices = count_if_v(handle,
                                   mg_graph_view,
                                   data,
                                   [] __device__(auto val) { return true; });
    std::cout<<"Count if vertices : "<<num_vertices<<"\n";


    // 3. compute max in-degree

    // 4. run MG Katz Centrality

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG Katz Centrality took " << elapsed_time * 1e-6 << " s.\n";
    }

    // 5. copmare SG & MG results

  }
};

using Tests_MG_CountIfV_File = Tests_MG_CountIfV<cugraph::test::File_Usecase>;
using Tests_MG_CountIfV_Rmat = Tests_MG_CountIfV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_CountIfV_File, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float>(GetParam());
}

TEST_P(Tests_MG_CountIfV_Rmat, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float>(GetParam());
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_CountIfV_File,
  ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx")));
 // ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
 //                   cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
 //                   cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
 //                   cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx")));

//INSTANTIATE_TEST_SUITE_P(rmat_small_test,
//                         Tests_MG_CountIfV_Rmat,
//                         ::testing::Values(cugraph::test::Rmat_Usecase(
//                             10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true)));

//INSTANTIATE_TEST_SUITE_P(rmat_large_test,
//                         Tests_MG_CountIfV_Rmat,
//                         ::testing::Values(cugraph::test::Rmat_Usecase(
//                             20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true)));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
