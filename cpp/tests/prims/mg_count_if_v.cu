#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/high_res_clock.h>
#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/partition_manager.hpp>

#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/prims/count_if_v.cuh>
#include <cuco/detail/hash_functions.cuh>

#include <raft/comms/comms.hpp>
#include <raft/comms/mpi_comms.hpp>
#include <raft/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <thrust/count.h>

#include <gtest/gtest.h>

#include <random>

// do the perf measurements
// enabled by command line parameter s'--perf'
//
static int PERF = 0;

template <typename input_usecase_t>
class Tests_MG_CountIfV
  : public ::testing::TestWithParam<input_usecase_t> {
 public:
  Tests_MG_CountIfV() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of count_if_v primitive and thrust count_if on a single GPU
  template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
  void run_current_test(input_usecase_t const &input_usecase)
  {
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto &comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) { --row_comm_size; }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }
    auto [mg_graph, d_mg_renumber_map_labels] =
      input_usecase.template construct_graph<vertex_t, edge_t, weight_t, true, true>(
        handle, true, true);

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();

    const int hash_bin_count = 5;

    auto primitive_lambda = [hash_bin_count] __device__(auto val) {
      cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
      return (0 == (hash_func(val) % hash_bin_count)); };

    // 4. run MG count if

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    vertex_t const * data = (*d_mg_renumber_map_labels).data();
    auto vertex_count = count_if_v(handle,
                                   mg_graph_view,
                                   data,
                                   primitive_lambda);

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG count if took " << elapsed_time * 1e-6 << " s.\n";
    }

    // 5. compare SG & MG results
    auto [sg_graph, d_sg_renumber_map_labels] =
      input_usecase.template construct_graph<vertex_t, edge_t, weight_t, true, false>(
        handle, true, false);
    auto sg_graph_view = sg_graph.view();
    auto expected_vertex_count =
      thrust::count_if(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                       thrust::make_counting_iterator(sg_graph_view.get_local_vertex_first()),
                       thrust::make_counting_iterator(sg_graph_view.get_local_vertex_last()),
                                                  primitive_lambda);
    ASSERT_TRUE(expected_vertex_count == vertex_count);
  }
};

using Tests_MG_CountIfV_File = Tests_MG_CountIfV<cugraph::test::File_Usecase>;
using Tests_MG_CountIfV_Rmat = Tests_MG_CountIfV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_CountIfV_File, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float>(GetParam());
}

TEST_P(Tests_MG_CountIfV_Rmat, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float>(GetParam());
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_CountIfV_File,
  ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                    cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                    cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                    cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx")));

INSTANTIATE_TEST_SUITE_P(rmat_small_test,
                         Tests_MG_CountIfV_Rmat,
                         ::testing::Values(cugraph::test::Rmat_Usecase(
                             10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true)));

INSTANTIATE_TEST_SUITE_P(rmat_large_test,
                         Tests_MG_CountIfV_Rmat,
                         ::testing::Values(cugraph::test::Rmat_Usecase(
                             20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true)));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
