#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/high_res_clock.h>
#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/partition_manager.hpp>

#include <cuco/detail/hash_functions.cuh>
#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/prims/reduce_v.cuh>

#include <thrust/count.h>
#include <raft/comms/comms.hpp>
#include <raft/comms/mpi_comms.hpp>
#include <raft/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <gtest/gtest.h>

#include <random>

// do the perf measurements
// enabled by command line parameter s'--perf'
//
static int PERF = 0;

template <typename vertex_t, typename result_t>
struct property_transform : public thrust::unary_function<vertex_t, result_t> {
  int mod{};
  property_transform(int mod_count) : mod(mod_count) {}
  __device__ result_t operator()(const vertex_t& val)
  {
    cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
    return static_cast<result_t>(hash_func(val) % mod);
  }
};

struct Prims_Usecase {
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_MG_ReduceIfV
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MG_ReduceIfV() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of reduce_if_v primitive and thrust reduce on a single GPU
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool store_transposed>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) {
      --row_comm_size;
    }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }
    auto [mg_graph, d_mg_renumber_map_labels] =
      input_usecase.template construct_graph<vertex_t, edge_t, weight_t, store_transposed, true>(
        handle, true, true);

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();

    const int hash_bin_count = 5;

    // 3. run MG count if

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    vertex_t const* data = (*d_mg_renumber_map_labels).data();
    rmm::device_uvector<result_t> test_property(d_mg_renumber_map_labels->size(),
                                                handle.get_stream());
    thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      data,
                      data + test_property.size(),
                      test_property.begin(),
                      property_transform<vertex_t, result_t>(hash_bin_count));
    auto vertex_count =
      reduce_v(handle, mg_graph_view, test_property.begin(), test_property.end(), result_t{0});

    if (PERF) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG count if took " << elapsed_time * 1e-6 << " s.\n";
    }

    // 4. compare SG & MG results

    if (prims_usecase.check_correctness) {
      cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, false> sg_graph(
        handle);
      std::tie(sg_graph, std::ignore) =
        input_usecase.template construct_graph<vertex_t, edge_t, weight_t, store_transposed, false>(
          handle, true, false);
      auto sg_graph_view         = sg_graph.view();
      auto expected_vertex_count = thrust::transform_reduce(
        rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
        thrust::make_counting_iterator(sg_graph_view.get_local_vertex_first()),
        thrust::make_counting_iterator(sg_graph_view.get_local_vertex_last()),
        property_transform<vertex_t, result_t>(hash_bin_count),
        result_t{0},
        thrust::plus<result_t>());
      ASSERT_TRUE(expected_vertex_count == vertex_count);
    }
  }
};

using Tests_MG_ReduceIfV_File = Tests_MG_ReduceIfV<cugraph::test::File_Usecase>;
using Tests_MG_ReduceIfV_Rmat = Tests_MG_ReduceIfV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_ReduceIfV_File, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_ReduceIfV_Rmat, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_ReduceIfV_File, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_ReduceIfV_Rmat, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_ReduceIfV_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MG_ReduceIfV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_large_test,
  Tests_MG_ReduceIfV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
